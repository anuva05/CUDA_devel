/************************************************
FILENAME: example3dcuda.cu

AUTHOR: Anuva K

DESCRIPTION: Transfer subvolume to GPU. Allocate memory for pencil signal.
Copy part of signal into the allocated signal. The rest is zero padding. Compute pencil FFT.
for now, let us consider 1d signal
*/



#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#define N_SIGS 16
#define IN_SIG_LEN 4
#define OUT_SIG_LEN 8
int main(){

  hipFloatComplex *h_signal, *h_pencil, *d_signal, *d_pencil, *h_result, *d_result;
  
  h_signal = (hipFloatComplex *)malloc(N_SIGS*IN_SIG_LEN*sizeof(hipFloatComplex));
  h_result = (hipFloatComplex *)malloc(N_SIGS*OUT_SIG_LEN*sizeof(hipFloatComplex));
  h_pencil = (hipFloatComplex *)malloc(N_SIGS*OUT_SIG_LEN*sizeof(hipFloatComplex));


  for (int i = 0; i < N_SIGS; i ++)
    for (int j = 0; j < IN_SIG_LEN; j++) // to include padding
     h_signal[(i*IN_SIG_LEN) + j] = make_hipFloatComplex(100*sin((i+1)*6.283*j/IN_SIG_LEN), 0); //this is how to put data into cuFloatComplex type variable


  hipMalloc(&d_signal, N_SIGS*IN_SIG_LEN*sizeof(hipFloatComplex));//same size as input subvolume
  hipMalloc(&d_pencil, N_SIGS*OUT_SIG_LEN*sizeof(hipFloatComplex));//allocate for pencil
  hipMalloc(&d_result, N_SIGS*OUT_SIG_LEN*sizeof(hipFloatComplex)); //size of full length pencil
  
  hipMemcpy(d_signal, h_signal, N_SIGS*IN_SIG_LEN*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

   for(int i=0; i< N_SIGS; i++)
    for(int j=0; j< OUT_SIG_LEN; j++)   
         h_pencil[(i*OUT_SIG_LEN)+j]=make_hipFloatComplex(0,0);
  
   hipMemcpy(d_pencil, h_pencil, N_SIGS*OUT_SIG_LEN*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

// COPY subvolume pencil into full pencil, for each
   for(int i=0; i< N_SIGS; i++)
    for(int j=0; j< IN_SIG_LEN; j++)  
//       d_pencil[(i*OUT_SIG_LEN)+j]= d_signal[(i*IN_SIG_LEN) + j];
       hipMemcpy(&d_pencil[(i*OUT_SIG_LEN)+j],&d_signal[(i*IN_SIG_LEN) + j], sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
  hipfftHandle plan;
  

  int n[1] = {OUT_SIG_LEN};

  hipfftResult res = hipfftPlanMany(&plan, 1, n,
     NULL, 1, OUT_SIG_LEN,  //advanced data layout, NULL shuts it off. idist=IN_SIG_LEN
     NULL, 1, OUT_SIG_LEN,  //advanced data layout, NULL shuts it off. odist= OUT_SIG_LEN
     HIPFFT_C2C, N_SIGS);
  if (res != HIPFFT_SUCCESS) {printf("plan create fail\n"); return 1;}

  res = hipfftExecC2C(plan, d_pencil, d_result, HIPFFT_FORWARD);



  if (res != HIPFFT_SUCCESS) {printf("forward transform fail\n"); return 1;}
  hipMemcpy(h_result, d_result, N_SIGS*OUT_SIG_LEN*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

  for (int i = 0; i < N_SIGS; i++){
    for (int j = 0; j < IN_SIG_LEN; j++)
      printf("%.3f ", hipCrealf(h_signal[(i*IN_SIG_LEN)+j]));
    printf("\n"); }

  printf("result:\n");


  for (int i = 0; i < N_SIGS; i++){
    for (int j = 0; j < OUT_SIG_LEN; j++)
      printf("%.3f ", hipCrealf(h_result[(i*OUT_SIG_LEN)+j]));
    printf("\n"); }

  return 0;
}
