/* DESCRIPTION: Trying small example to test whether array is transposed on GPU 
side while 3d CUFFT is being performed dimension wise.
Result of DFT on x-dimension is compared with transposed DFT on host side.
Playing around with hipfftPlanMany.
Computes DFT of each ROW of the matrix. Hence it computes N DFTs of size M.
*/


#include<stdio.h>
#include<hipfft/hipfft.h>
#include<hip/hip_complex.h>
#define N 4
#define M 2
#define RANK 1 //only 1d transforms are being computed
int main(){

hipFloatComplex *d_signal, *d_result, *h_signal, *h_result;
int BATCH 4 //the "howmany" parameter in fftw

h_signal= (hipFloatComplex *)malloc(N*M*sizeof(hipFloatComplex));
h_result= (hipFloatComplex *)malloc(N*M*sizeof(hipFloatComplex));

hipMalloc(&d_signal, N*M*sizeof(hipFloatComplex));
hipMalloc(&d_result, N*M*sizeof(hipFloatComplex));


for(int i=0; i<N; i++)
 for(int j=0; j<M; j++)
   h_signal[i*M + j] =make_hipFloatComplex( i*M + j,0);

hipMemcpy(d_signal,h_signal,N*M*sizeof(hipFloatComplex),hipMemcpyHostToDevice);


//parameters for the "many" transform
int istride, ostride, idist, odist;
hipfftHandle plan;


int n[]={2}; //1d transforms of length M
istride =1;
ostride=1;
idist= M;
odist = M;
int *iembed= n, *oembed=n;


hipfftPlanMany(&plan, RANK, n, iembed, istride, idist,  oembed, ostride, odist, HIPFFT_C2C, BATCH);
    
 hipfftExecC2C(plan, d_signal, d_result, HIPFFT_FORWARD);
 hipDeviceSynchronize();
  
 hipfftDestroy(plan);
 hipFree(d_signal);

 hipMemcpy(h_result, d_result, N*M*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++){
    for (int j = 0; j < M; j++)
      printf("%.3f ", hipCrealf(h_signal[(i*M)+j]));
    printf("\n"); }

  printf("result:\n");


  for (int i = 0; i < N; i++){
    for (int j = 0; j < M; j++)
      printf("%.3f ", hipCrealf(h_result[(i*M)+j]));
    printf("\n"); }

  return 0;



}
