/************************************************
FILENAME: example_paddedpencil.cu

AUTHOR: Anuva K

DESCRIPTION: Test code to perform 3d FFTs on CUDA
according to the proposed pruned framework. FFTs of a small
non-zero subvolume of a larger volume of zeros are to be 
computed pencil by pencil without storing the large 3d array
This script tests X dimension FFT of k x k x k signal to N x k x k
*/ 




#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#define N_SIGS 1 
#define IN_SIG_LEN 8
#define OUT_SIG_LEN 8
int main(){

  hipFloatComplex *h_signal, *d_signal, *h_result, *d_result;

  h_signal = (hipFloatComplex *)malloc(N_SIGS*IN_SIG_LEN*sizeof(hipFloatComplex));
  h_result = (hipFloatComplex *)malloc(N_SIGS*OUT_SIG_LEN*sizeof(hipFloatComplex));
  for (int i = 0; i < N_SIGS; i ++)
    for (int j = 0; j < IN_SIG_LEN/2; j++) // to include padding
  h_signal[(i*IN_SIG_LEN) + j] = make_hipFloatComplex(100*sin((i+1)*6.283*j/IN_SIG_LEN), 0); //this is how to put data into cuFloatComplex type variable
  hipMalloc(&d_signal, N_SIGS*IN_SIG_LEN*sizeof(hipFloatComplex));
  hipMalloc(&d_result, N_SIGS*OUT_SIG_LEN*sizeof(hipFloatComplex));

  hipMemcpy(d_signal, h_signal, N_SIGS*IN_SIG_LEN*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  hipfftHandle plan;
  int n[1] = {IN_SIG_LEN};

  hipfftResult res = hipfftPlanMany(&plan, 1, n,
     NULL, 1, IN_SIG_LEN,  //advanced data layout, NULL shuts it off. idist=IN_SIG_LEN
     NULL, 1, OUT_SIG_LEN,  //advanced data layout, NULL shuts it off. odist= OUT_SIG_LEN
     HIPFFT_C2C, N_SIGS);
  if (res != HIPFFT_SUCCESS) {printf("plan create fail\n"); return 1;}

  res = hipfftExecC2C(plan, d_signal, d_result, HIPFFT_FORWARD);
  if (res != HIPFFT_SUCCESS) {printf("forward transform fail\n"); return 1;}
  hipMemcpy(h_result, d_result, N_SIGS*OUT_SIG_LEN*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

  for (int i = 0; i < N_SIGS; i++){
    for (int j = 0; j < IN_SIG_LEN; j++)
      printf("%.3f ", hipCrealf(h_signal[(i*IN_SIG_LEN)+j]));
    printf("\n"); }

  printf("result:\n");


  for (int i = 0; i < N_SIGS; i++){
    for (int j = 0; j < OUT_SIG_LEN; j++)
      printf("%.3f ", hipCrealf(h_result[(i*OUT_SIG_LEN)+j]));
    printf("\n"); }

  return 0;
}
