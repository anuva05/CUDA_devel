#include<stdio.h>
#include<hipfft/hipfft.h>
#include <hip/hip_complex.h>
#define BATCH 1
int main(){


   // Transform each column of a 2d array with 10 rows and 3 columns:

  //int rank = 1; /* not 2: we are computing 1d transforms */
  //int n[] = {10}; /* 1d transforms of length 10 */
  //int howmany = 3;
  // int idist = odist = 1;
  // int istride = ostride = 3; /* distance between two elements in 
  //
  //
  //                     
  //           the same column */
   int n[]={10};
   int *iembed = n, *oembed = n;  
   int rank=1;
   int howmany = 3;
   int idist=1,odist= 1;
   int istride=3,ostride=3;
   hipfftComplex** datacpu[10][3], data[10][3];   

   datacpu= (hipfftComplex **)malloc(sizeof(hipfftComplex)*10*3);
   datacpu[0][1]=6;
   datacpu[1][3]=7;
   
   

    hipfftHandle plan;
    
     hipMalloc((void**)&data, sizeof(hipfftComplex)*10*3);
    hipMemcpy(datacpu,data, 10*3*sizeof(hipfftComplex), hipMemcpyHostToDevice);
    hipfftPlanMany(&plan, rank, n, &iembed, istride, idist,  &oembed, ostride, odist, HIPFFT_C2C, BATCH);
    


    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    





    //free memory
    hipfftDestroy(plan);
    hipFree(data);
    printf("fin");
    return 0;
}

