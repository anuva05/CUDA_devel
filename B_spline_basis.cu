#include "hip/hip_runtime.h"
#include<stdio.h>

__global__  void generate_basis(double x,int i,double *knots,int n){

// x =  1d point
// i =augmented knot index
//knots= 
//n = degree of B spline

int m = n+1;
int N= sizeof(knots);
double alpha1, alpha2;
float *B[N+2*m][n+1];


	if(n==0){

         if(knots[i]<=x) && (x<knots[i+1]){
         B[i][n]=1 ;}
         else{
         B[i][n]=0;
	}
        return(B);
 	}//if n=0

       
       else {
	if((knots[n+i] - knots[i]) == 0) {
		alpha1 = 0;
		} else {
	 	alpha1 = (x - knots[i])/(knots[n+i] - knots[i]);
                 }
	if((knots[i+n+1] - knots[i+1]) == 0) {
	alpha2 = 0;
	} else 
	alpha2 = (knots[i+n+1] - x)/(knots[i+n+1] - knots[i+1]);
}
	B = alpha1*generate_basis(x, (n-1), i, knots) + alpha2*generate_basis(x, (n-1), (i+1), knots);
}
	return(B);



}
